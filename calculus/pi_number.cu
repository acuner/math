#include<stdio.h>
#include<hip/hip_runtime.h>


#define BLOCK_NUM 32   //块数量
#define THREAD_NUM 256 // 每个块中的线程数
#define LOOP_N BLOCK_NUM * THREAD_NUM * 1000000

__global__ void pi2(double* result) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    double tmp = 0;
    int flag = -1;
    int idx = bid * THREAD_NUM + tid;
    int start = idx * 100000 + 1;
    int end = start + 100000;
    for (int i = start; i < end; i++) {
        tmp += flag * (1./(2 * i + 1));
        flag = -flag;
    }
    result[bid*THREAD_NUM+tid] = tmp;
}

int main(){
    double *sum, *result;
    double pi_v = 1;

    // allocate host memory
    sum = (double*) malloc(sizeof(double) * BLOCK_NUM * THREAD_NUM);

    // Allocate device memory
    hipMalloc((void **)&result, sizeof(double) * BLOCK_NUM * THREAD_NUM);
    
    // Execute kernels
    pi2<<<BLOCK_NUM,THREAD_NUM>>>(result);

    // Transfer output from device memory to host
    hipMemcpy(sum, result, sizeof(double)*BLOCK_NUM*THREAD_NUM, hipMemcpyDeviceToHost);

    for (int i = 0; i < BLOCK_NUM * THREAD_NUM; i++) {
        pi_v += sum[i];
    }

    printf("calculate %.10f\n", pi_v*4);

    hipFree(result);
    free(sum);
}
