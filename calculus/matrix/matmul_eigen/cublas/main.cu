#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"


void print_matrix(float* mat, int row, int col) {
    for (int i = 0; i < row * col; i++) {
        printf("%f\t", mat[i]);
        if ((i+1) % col == 0) {
            printf("\n");
        }
        
    }
     printf("----------------------------\n");
}


int main(int argc, char *argv[]) {
    float *mat1, *mat2, *result;
    float *g_mat1, *g_mat2, *g_mat_result;
    int r_size, m_size;    // 矩阵行数，矩阵size

    hipError_t cudaStat;
    hipblasHandle_t handle;
    hipblasStatus_t stat;

    if (argc > 1) {
        r_size = atoi(argv[1]);
    } else {
        r_size = 8192;
    }
    m_size = r_size * r_size;
    
    // 用一位数组表示二维矩阵
    mat1 = (float*) malloc(m_size * sizeof(float));
    mat2 = (float*) malloc(m_size * sizeof(float));
    result = (float*) malloc(m_size * sizeof(float));

    // initialize
    for (int i = 0; i < m_size; i++) {
        mat1[i] = rand()/10000000;
        mat2[i] = rand()/10000000;
        result[i] = 0; 
    }

    cudaStat = hipMalloc((void **)&g_mat1, sizeof(*mat1) * m_size);
    cudaStat = hipMalloc((void **)&g_mat2, sizeof(*mat2) * m_size);
    cudaStat = hipMalloc((void **)&g_mat_result, sizeof(*result) * m_size);
    printf("cudaStat %d\n", cudaStat);

    // initialize CUBLAS context
    stat = hipblasCreate(&handle);

    stat = hipblasSetMatrix(r_size, r_size, sizeof(*mat1), mat1, r_size, g_mat1, r_size);
    stat = hipblasSetMatrix(r_size, r_size, sizeof(*mat2), mat2, r_size, g_mat2, r_size);
    stat = hipblasSetMatrix(r_size, r_size, sizeof(*result), result, r_size, g_mat_result, r_size);

    float al = 1.0f;
    float bet = 0.0f;
    
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        r_size, r_size, r_size, &al, g_mat1, 
        r_size, g_mat2, r_size, &bet, g_mat_result, r_size);
    stat = hipblasGetMatrix(r_size, r_size, sizeof(*result), g_mat_result, r_size, result, r_size);
    printf("cublas %d\n",stat);
    // cudaMemcpy(result, g_mat_result, sizeof(float) * m_size, cudaMemcpyDeviceToHost);
    
    if (r_size < 10) {
        printf("-----mat1----\n");
        print_matrix(mat1, r_size, r_size);
        printf("-----mat2----\n");
        print_matrix(mat2, r_size, r_size);
        printf("----mat1 * mat2---\n");
        print_matrix(result, r_size, r_size);
    }
    hipFree(g_mat1);
    hipFree(g_mat2);
    hipFree(g_mat_result);
    free(mat1);
    free(mat2);
    free(result);
}